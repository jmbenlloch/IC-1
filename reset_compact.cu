
#include <hip/hip_runtime.h>
#include <stdio.h>

struct voxel{
	float x;
	float y;
	float E;
};

// due to packing this is 12 bytes instead of 9, to be changed
struct sensor{
	int id;
	float charge;
	bool active;
};


// Launch: block < (xmax - xmin)/xsize, (ymax - ymin)/ysize >, grid <1,1>
// All voxels are initialized with "charge" and only those within 
// fiducial volume are set to be active

// Has to be one block and not one grid. We loose synchronization.
// Maximum 2k threads. We have up to ~30k voxels. How to map them to threads?
__global__ void create_voxels_compact(voxel * voxels, int * address,
		bool * actives, float xmin, float xmax, float ymin, float ymax, 
		float xsize, float ysize, float rmax, float charge){

	int offset = blockIdx.x * blockDim.x;
	float x = xmin + xsize * blockIdx.x;
	float y = ymin + ysize * threadIdx.x;
	bool active = sqrtf(x*x + y*y) < rmax;

	// Mem layout: voxels[x][y]
	int pos = offset + threadIdx.x;
	address[pos] = active;
	actives[pos] = active;
//	printf("[%d, %d] address[%d]: %d\n", blockIdx.x, threadIdx.x, pos, address[pos]);
	__syncthreads();

	// Scan algoritm (Hillis-Steele)
	for(int idx=1; idx <= threadIdx.x; idx <<= 1){
		int new_value = address[pos] + address[pos - idx];
		__syncthreads();
		address[pos] = new_value;
//		printf("#########\n");
//		printf("[%d, %d] value: %d, idx: %d, pos: %d\n" , blockIdx.x, threadIdx.x, address[pos], idx, pos);
	}

	//Write active voxels in their address
	// Addresses are shifted 1 position due to scan algorithm
	if(active){
		voxel * v = voxels + offset + address[pos] - 1;
		v->x = x;
		v->y = y;
		v->E = charge;
	}
}


// Launch block < xdim, 1, 1>, grid <1,1>
__global__ void compact_voxels(voxel * voxels, voxel * voxels_compact,
		int * address, bool * actives, int ydim){
	extern __shared__ int offset[];

	int pos = threadIdx.x * ydim - 1;
	if(threadIdx.x == 0){
		pos += 1;
	}

//	printf("[%d]: address: %d\n", threadIdx.x, pos);

	offset[threadIdx.x] = address[pos];
//	printf("[%d]: %d\n", threadIdx.x, offset[threadIdx.x]);

	//Scan offset vector
	for(int idx=1; idx <= threadIdx.x; idx<<=1){
		int value = offset[threadIdx.x] + offset[threadIdx.x - idx];
		__syncthreads();
		offset[threadIdx.x] = value;

//		printf("-[%d]: idx: %d, value: %d\n", threadIdx.x, idx, offset[threadIdx.x]);
	}

//	printf("scan [%d]: %d\n", threadIdx.x, offset[threadIdx.x]);

	// Compact vector
	for(int i=0; i<ydim; i++){
		int offset_in = threadIdx.x * ydim + i;
//		printf("[%d]: offset_in: %d, active: %d\n", threadIdx.x, offset_in, actives[offset_in]);
		if(actives[offset_in]){
			voxel * v_out = voxels_compact + offset[threadIdx.x] - 1 + i;
			voxel * v_in  = voxels + offset_in;
//			printf("[%d]: compact: %d\tnon_compact: %d\n", threadIdx.x, offset[threadIdx.x] - 1 + i, threadIdx.x + i);
			v_out->x = v_in->x;
			v_out->y = v_in->y;
			v_out->E = v_in->E;
		}
	}
}

// Launch <<< nsensors = 1792 >>>
__global__ void initialize_anode(sensor * sensors, float xmin, float xmax, float * xs, float ymin, float ymax, float * ys, float sipm_dist){
	sensor * s = sensors + blockIdx.x;
	int id = blockIdx.x;

	s->active = xs[id] > (xmin - sipm_dist) && xs[id] < (xmax + sipm_dist) &&
		ys[id] > (ymin - sipm_dist) && ys[id] < (ymax + sipm_dist);
	s->charge = 0;
	s->id = id; 

	//printf("[%d]: id=%d, charge=%f, active=%d\n", blockIdx.x, s->id, s->charge, s->active);
}

// Launch < #sensors in slice >
__global__ void create_anode_response(sensor * sensors, int * ids, float * charges){
	int id = ids[blockIdx.x];
	sensor * s = sensors + id; 
	s->charge = charges[blockIdx.x];

	//  printf("[%d]: id=%d, %d, charge=%f, active=%d\n", blockIdx.x, id, s->id, s->charge, s->active);
}

