
#include <hip/hip_runtime.h>
#include <stdio.h>

struct voxel{
	float x;
	float y;
	float E;
};

struct segmented_scan{
	float value;
	int active;
};

struct mlem_scan{
	float eff;
	float projection;
	int flag;
};

struct correction{
	float x;
	float y;
	float factor;
};

// Launch: block <1024, 1, 1>, grid <nslices,1>
// All voxels are initialized with "charge" and only those within 
// fiducial volume are set to be active
__global__ void create_voxels(voxel * voxels, 
		int * slice_start, float * xmins, float * xmaxs, 
		float * ymins, float * ymaxs, float * charges, float xsize, 
		float ysize, float rmax, bool * actives, int * address){

	int offset = slice_start[blockIdx.x];
	float charge = charges[blockIdx.x];
	if(threadIdx.x == 0){
//		printf("[%d][%d]: start: %d\n", blockIdx.x, threadIdx.x, offset);
	}

	int xmin = xmins[blockIdx.x];
	int xmax = xmaxs[blockIdx.x];
	int ymin = ymins[blockIdx.x];
	int ymax = ymaxs[blockIdx.x];
	int xsteps = (xmax - xmin) / xsize;
	int ysteps = (ymax - ymin) / ysize;

	int iterations = ceilf(1.f*(xsteps*ysteps)/blockDim.x);
//	printf("iterations: %d\n", iterations);

	for(int i=0; i<iterations; i++){
		int vid = threadIdx.x + i*blockDim.x;
		float x = xmin + (vid / ysteps) * xsize;
		float y = ymin + (vid % ysteps) * xsize;

		//TODO Check boundary condition
		if(x < xmax && y < ymax){
			bool active = sqrtf(x*x + y*y) < rmax;
			voxel * v = voxels + offset + vid;
//			printf("[%d][%d][%d]: offset %d, vid %d\n", blockIdx.x, threadIdx.x, i, offset, vid);
			v->x = x;
			v->y = y;
			v->E = charge;
			//printf("[%d][%d][%d]: pos: (%f, %f), steps: (%d, %d)\n", blockIdx.x, threadIdx.x, i, x, y, xsteps, ysteps);
//			printf("[%d][%d][%d]: pos: (%f, %f), steps: (%d, %d)\n", blockIdx.x, threadIdx.x, i, v->x, v->y, xsteps, ysteps);
			actives[offset + vid] = active;
			address[offset + vid] = active;
		}
	}
}


// Launch block < xdim, 1, 1>, grid <1,1>
__global__ void compact_voxels(voxel * voxels_nc, voxel * voxels,
		int * address, bool * actives, int * slice_start_nc, int * slice_start){
	int start = slice_start_nc[blockIdx.x];
	int end   = slice_start_nc[blockIdx.x+1];
	int steps = end - start;
	int iterations = ceilf(1.f*steps/blockDim.x);
	if(threadIdx.x == 0){
//		printf("[%d], start %d, end %d, steps %d, iterations: %d\n", blockDim.x, start, end, steps, iterations);
		slice_start[blockIdx.x] = address[start] - 1;
		if (blockIdx.x == 0){
			int lastSlice = slice_start_nc[gridDim.x]-1;
			slice_start[gridDim.x] = address[lastSlice];
		}
	}

	// Compact vector
	for(int i=0; i<iterations; i++){
		int vidx = threadIdx.x + i*blockDim.x;
		int offset = start + vidx;
		if(offset < end && actives[offset]){
			voxel * v_out = voxels + address[offset] - 1;
			voxel * v_in  = voxels_nc + offset;
//			printf("[%d]: offset %d, out %d\n", blockIdx.x, offset, address[offset]);
			v_out->x = v_in->x;
			v_out->y = v_in->y;
			v_out->E = v_in->E;
		}   
	}   
}

// Launch grid <nslices, 1>, block <1024, 1, 1>
__global__ void create_anode_response(float * anode_response, int nsensors,
		int * sensors_ids, float * charges, int * slices_start){
	int start = slices_start[blockIdx.x];
	int end   = slices_start[blockIdx.x+1];
	int steps = end - start;
	int offset = nsensors * blockIdx.x;
	int iterations = ceilf(1.f*steps/blockDim.x);
//	printf("[%d]: iterations=%d, start=%d, end=%d, steps=%d, offset=%d\n", blockIdx.x, iterations, start, end, steps, offset);

	for(int i=0; i<iterations; i++){
		int step = threadIdx.x + i*blockDim.x;
		int sidx = start + step;
		if (step < steps){
			int sensor_pos = offset + sensors_ids[sidx];
//			printf("[%d]: iterations=%d, sidx=%d, sid=%d, charge=%f, pos=%d\n", blockIdx.x, iterations, sidx, sensors_ids[sidx], charges[sidx], sensor_pos);
			anode_response[sensor_pos] = charges[sidx];
		}
	}
}


// Launch block <1024,1,1>, grid < ceil(nvoxels/1024), 1>
__global__ void compute_active_sensors(float * probs, bool * active, int * address,
		int nvoxels, int nsensors, int sensors_per_voxel, voxel * voxels, float sensor_dist, 
		float * xs, float * ys, float step, int nbins, float xmin, float ymin,
	   	correction * corrections){
	int vidx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("[%d][%d] id: %d, nsensors: %d\n", blockIdx.x, threadIdx.x, vidx, nsensors);

	int base_idx = vidx * sensors_per_voxel;
	int active_count = 0;

	//Check bounds
	if(vidx < nvoxels){
		for(int sidx=0; sidx<nsensors; sidx++){
			int idx = base_idx + active_count;
			float xdist = voxels[vidx].x - xs[sidx];
			float ydist = voxels[vidx].y - ys[sidx];

			bool voxel_sensor = ((abs(xdist) <= sensor_dist) &&
					(abs(ydist) <= sensor_dist));
			active_count += voxel_sensor;

			//Compute index
			active[idx]  = voxel_sensor;
			address[idx] = voxel_sensor;

			// Compute probability
			// In order to avoid accesing wrong parts of the memory 
			// if the sensor is not active for a particular voxel,
			// then we will use index 0.
			// Rounding: plus 0.5 and round down
			int xindex = __float2int_rd((xdist - xmin) / step * voxel_sensor + 0.5f);
			int yindex = __float2int_rd((ydist - ymin) / step * voxel_sensor + 0.5f);
			int prob_idx = xindex * nbins + yindex;

			probs[idx] = corrections[prob_idx].factor;
		}
	}
}
